//
// TP: Lancer un kernel vide sur le GPU
//


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void emptyKernel ();

int main(int argc, char** argv) {

  hipSetDevice(0);

  emptyKernel<<<1,1>>>();

  hipDeviceSynchronize();
  std::cout << "Hello, CUDA!" << std::endl; 
  return 0;
}

__global__ void emptyKernel(){
  // Empty
}
